#include "hip/hip_runtime.h"
/**
@file degnome.c
@page degnome
@author Daniel R. Tabin
@brief Digital Genomes aka Degnomes

This program will be used to simulated Polygenic evoltion of
quantitative traits by using Degnomes as defined above.
*/
#include "cuda_degnome.h"
#include "fitfunc.h"
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
	void unscramble_generation(int blocksCount, int threadsCount, Degnome* source, Degnome* dest, int num_ranks, int sub_pop_size, int chrom_size);
	void Degnome_reorganize(size_t blocksCount, size_t threadsCount, Degnome* q, int pop_size, int chrom_size);
	Degnome* Degnome_cuda_new(int pop_size, int chrom_size);
	// void Degnome_mate(Degnome* child, Degnome* p1, Degnome* p2, hiprandStateXORWOW_t* state,
	// 				int mutation_rate, int mutation_effect, int crossover_rate, int chrom_size);
	void Degnome_cuda_free(Degnome* q);
}

// void cuda_byte_copy(void) {

// }

void memory_copy(char* source, char* dest, int length) {
	for (int i = 0; i < length; i++) {
		dest[i] = source[i];
	}
}

// void unscramble_generation(int blocksCount, int threadsCount, Degnome* source, Degnome* dest, int num_ranks, int sub_pop_size, int chrom_size) {

// 	int full_pop_size = (sub_pop_size*num_ranks);
// 	printf("full pop size%u\n", full_pop_size);

// 	Degnome_reorganize(blocksCount, threadsCount, dest, full_pop_size, chrom_size);

// 	printf("reorganized\n");

// 	for (int i = 0; i < full_pop_size; i++) {
// 		printf("filling %u\n", i);
// 		dest[i].hat_size = 0;

// 		for (int j = 0; j < chrom_size; j++) {
// 			dest[i].dna_array[j] = 0;
// 		}
// 		dest[i].fitness = 0;
// 	}
// 	printf("filled\n");

// }

void unscramble_generation(int blocksCount, int threadsCount, Degnome* source, Degnome* dest, int num_ranks, int sub_pop_size, int chrom_size) {


	printf("Source DNA 0 %lf\n", ((double*) (source+sub_pop_size))[0]);

	printf("start\n");
	printf("%u\n", (num_ranks*sub_pop_size));
	Degnome* dest_end_of_dengomes = dest + (num_ranks*sub_pop_size);

	double* dest_DNA = (double*) dest_end_of_dengomes;
	Degnome* dest_Degnomes = dest;

	Degnome* source_Degnomes = source;
	Degnome* Degnome_converter = source + sub_pop_size;
	double* source_DNA = (double*) Degnome_converter;

	double* double_converter = NULL;

	printf("done init\n");

	for (int i = 0; i < num_ranks; i++) {

		printf("Source hat_size %lf\n", source_Degnomes->hat_size);
		printf("Source DNA 0 %lf\n", source_DNA[0]);
		printf("copying, %u\n", i);
		memory_copy(((char*) source_Degnomes), ((char*) dest_Degnomes), (sub_pop_size*sizeof(Degnome)));
		memory_copy(((char*) source_DNA), ((char*) dest_DNA), (sub_pop_size*chrom_size*sizeof(double)));

		printf("Dest hat_size %lf\n", dest_Degnomes->hat_size);
		printf("Dest DNA 0 %lf\n", dest_DNA[0]);

		printf("done copying, %u\n", i);

		dest_DNA += (sub_pop_size*chrom_size);
		dest_Degnomes += sub_pop_size;

		printf("updated dest, %u\n", i);

		source_DNA += (sub_pop_size*chrom_size);
		source_Degnomes += sub_pop_size;

		double_converter = (double*) source_Degnomes;
		Degnome_converter = (Degnome*) source_DNA;

		double_converter += (sub_pop_size*chrom_size);
		Degnome_converter += sub_pop_size;

		source_DNA = (double*) Degnome_converter;
		source_Degnomes = (Degnome*) double_converter;


		printf("updated source, %u\n", i);
	}	
}

__global__ void kernel_regorganize(Degnome* q, int pop_size, int chrom_size) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	Degnome* end_of_dengomes = q + pop_size;
	double* ptr_itr = (double*) end_of_dengomes;

	while (index < pop_size) {
		// printf("Index: %u\n", index);
		// set the dna array
		q[index].dna_array = ptr_itr + (index * chrom_size);
		// move the pointer
		index += blockDim.x * gridDim.x;
	}
}

void Degnome_reorganize(size_t blocksCount, size_t threadsCount, Degnome* q, int pop_size, int chrom_size) {
	kernel_regorganize<<<blocksCount,threadsCount>>>(q, pop_size, chrom_size);

	hipDeviceSynchronize();
}

Degnome* Degnome_cuda_new(int pop_size, int chrom_size) {
	Degnome* q;

	// calculate the size of a degnmed based on the chromosome length
	int degnome_size = (sizeof(Degnome) + (chrom_size * sizeof(double)));

	// malloc a single chunk of memory for easy MPI transfer
	hipMallocManaged(&q, (pop_size*degnome_size));

	return q;
}

// // device function
// __device__ void Degnome_mate(Degnome* child, Degnome* p1, Degnome* p2, void* rng_ptr,
// 	int mutation_rate, int mutation_effect, int crossover_rate, int chrom_size) {
// 	// printf("mating\n");

// 	//get rng
// 	hiprandStateXORWOW_t* state = (hiprandStateXORWOW_t*) rng_ptr;
	
// 	//Cross over
// 	int num_crossover = hiprand_poisson(state, crossover_rate);
// 	int crossover_locations[num_crossover];
// 	int distance = 0;
// 	int diff;

// 	for (int i = 0; i < num_crossover; i++) {
// 		crossover_locations[i] = (hiprand_poisson(state) % chrom_size);
// 	}
// 	if (num_crossover > 0) {
// 		int_qsort(crossover_locations, num_crossover);//changed
// 	}

// 	for (int i = 0; i < num_crossover; i++) {
// 		diff = crossover_locations[i] - distance;

// 		if (i % 2 == 0) {
// 			hipMemcpy(child->dna_array+distance, p1->dna_array+distance, (diff*sizeof(double)), hipMemcpyDefault);
// 		}
// 		else {
// 			hipMemcpy(child->dna_array+distance, p2->dna_array+distance, (diff*sizeof(double)), hipMemcpyDefault);
// 		}
// 		distance = crossover_locations[i];
// 	}

// 	if (num_crossover > 0) {
// 		diff = chrom_size - crossover_locations[num_crossover-1];
// 	}
// 	else {
// 		diff = chrom_size;
// 	}

// 	if (i % 2 == 0) {
// 		hipMemcpy(child->dna_array+distance, p1->dna_array+distance, (diff*sizeof(double)), hipMemcpyDefault);
// 	}
// 	else {
// 		hipMemcpy(child->dna_array+distance, p2->dna_array+distance, (diff*sizeof(double)), hipMemcpyDefault);
// 	}

// 	child->hat_size = 0;

// 	//mutate
// 	double mutation;
// 	int num_mutations = hiprand_poisson(state, mutation_rate);
// 	int mutation_location;

// 	for (int i = 0; i < num_mutations; i++) {
// 		mutation_location = (hiprand_poisson(state) % chrom_size);
// 		mutation = (hiprand_normal_double(state) * mutation_effect);
// 		child->dna_array[mutation_location] += mutation;
// 	}

// 	//calculate hat_size

// 	for (int i = 0; i < chrom_size; i++) {
// 		child->hat_size += child->dna_array[i];
// 	}

// 	// calculate fitness via cuda

// 	child->fitness = get_fitness(child->hat_size);
// 	//and we are done!
// }

void Degnome_cuda_free(Degnome* q) {
	// no need to free the dna_array as it is part of q
	// I think, we may want to test this
	hipFree(q);
}